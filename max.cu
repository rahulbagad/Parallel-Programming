
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void maxi(int *a,int *ans)
{

        int tID = blockIdx.x;
        if(tID<10)
        {

                if(a[tID]>*ans)
                {
                     *ans=a[tID];

                }

        }

}


int main()
{
        int a[10],i;
        int c=0;
        int *da,*ans;
        hipMalloc((void **) &da, 10*sizeof(int));
        hipMalloc((void **) &ans, 1*sizeof(int));
        printf("dya");
        for(i=0;i<10;i++)
        {
                scanf("%d",&a[i]);

        }
        printf("var");
        hipMemcpy(da, a, 10*sizeof(int), hipMemcpyHostToDevice);
        printf("khali");
        hipMemcpy(ans,&a[0],1*sizeof(int), hipMemcpyHostToDevice);

        maxi<<<10,1>>>(da,ans);
        hipMemcpy(&c, ans, sizeof(int), hipMemcpyDeviceToHost);
        printf("\nmax:%d",c);

        return 0;
}

