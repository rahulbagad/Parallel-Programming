#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define SIZE 3

__global__ void max(int *a , int *c )
{
extern __shared__ int sdata[];

unsigned int tid = threadIdx.x;
unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

sdata[tid] = a[i];

__syncthreads();

///////////////////////////
for(unsigned int s=1; s<blockDim.x; s*=2)
{
int index = 2 * s * tid;
if(index < blockDim.x)
{
sdata[index] += sdata[index + s];
}
//////////////////////////////
__syncthreads();
}
if(tid == 0) c[blockIdx.x] = sdata[0];

}
//////////////////////////
int main()
{
int i;
srand(time(NULL));

int *a;
a = (int*)malloc(SIZE * sizeof(int));
int c;

int *dev_a, *dev_c;

hipMalloc((void **) &dev_a, SIZE*sizeof(int));
hipMalloc((void **) &dev_c, SIZE*sizeof(int));

for( i = 0 ; i < SIZE ; i++)
{
a[i] = rand()% 20 + 1;
}
for( i = 0 ; i < SIZE ; i++)
{
printf("%d ",a[i]);
}

hipMemcpy(dev_a , a, SIZE*sizeof(int),hipMemcpyHostToDevice);
max<<<1,SIZE>>>(dev_a,dev_c);
hipMemcpy(&c, dev_c, SIZE*sizeof(int),hipMemcpyDeviceToHost);

printf("
sum =  %d 
",c);

hipFree(dev_a);
hipFree(dev_c);

printf("
");

return 0;
}
#4

