
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void add(int *a,int *b,int *c)
{
	int tID = blockIdx.x;
	if(tID<10)
	{
		c[tID]=a[tID]+b[tID];
	}
}


int main()
{
	int a[10],b[10],c[10],i;
	int *da,*db,*dc;
	hipMalloc((void **) &da, 10*sizeof(int));
	hipMalloc((void **) &db, 10*sizeof(int));
	hipMalloc((void **) &dc, 10*sizeof(int));
	for(i=0;i<10;i++)
	{
		a[i]=i;
		b[i]=2*i;
	}
	

	hipMemcpy(da, a, 10*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, b, 10*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dc, c, 10*sizeof(int), hipMemcpyHostToDevice);

	add<<<10,1>>>(da, db, dc);
	hipMemcpy(c, dc, 10*sizeof(int), hipMemcpyDeviceToHost);
	
	for(i=0;i<10;i++)
	{
		printf("%d + %d = %d\n",a[i],b[i],c[i]);
	}
	return 0;
}
